#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"
#include "matmul_gpu_basic.h"

/* matmul的函数实现*/
__global__ void MatmulKernel(float *M_device, float *N_device, float *P_device, int width){
    /* 
        我们设定每一个thread负责P中的一个坐标的matmul
        所以一共有width * width个thread并行处理P的计算
    */
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float P_element = 0;
    for (int k = 0; k < width; k++){
        float M_element = M_device[y * width + k]; // 行
        float N_element = N_device[k * width + x]; // 列
        P_element += M_element * N_element;  // 这个结束就是行列相乘
    }

    P_device[y * width + x] = P_element; // 第几行 + 第几列
}

/*
    CUDA中使用block对矩阵中某一片区域进行集中计算。这个类似于loop中的tile
    感兴趣的同学可以试着改一下blockSize，也就是tileSize，看看速度会发生什么样子的变化
    当blockSize达到一个数量的时候，这个程序会出错。下一个案例中我们会分析
*/
void MatmulOnDevice(float *M_host, float *N_host, 
                    float* P_host, int width, int blockSize)
{
    /*
    M_host: First Matrix ptr at host 
    h_host: second matrix ptr at host
    P_host: output matrix ptr at host 
    */
   // 设置矩阵尺寸
    int size = width * width* sizeof(float);
    // 开辟GPU内存
    float *M_device;
    float *N_device;
    float *P_device;

    hipMalloc(&M_device, size);
    hipMalloc(&N_device, size);
    hipMalloc(&P_device, size);

    // 把输入输出的矩阵信息从host搬到device
    hipMemcpy(M_device, M_host, size, hipMemcpyHostToDevice);
    hipMemcpy(N_device, N_host,  size, hipMemcpyHostToDevice);

    // 分配grid, block
    dim3 dimBlock(blockSize, blockSize);
    int gridDim = (width + blockSize - 1) / blockSize;
    dim3 dimGrid(gridDim, gridDim);

    // 调用kernel function计算
    MatmulKernel<<<dimGrid, dimBlock>>>(M_device, N_device, P_device, width);

    // 计算结果从device搬到host
    hipMemcpy(P_host, P_device, size , hipMemcpyDeviceToHost);

    // 等待全部线程完成计算
    hipDeviceSynchronize();

    // Free
    hipFree(P_device);
    hipFree(M_device);
    hipFree(N_device);

}

